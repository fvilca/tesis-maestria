#include "hip/hip_runtime.h"
#include <thrust/binary_search.h>
#include <thrust/fill.h>
#include <thrust/copy.h>
#include <thrust/count.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/extrema.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>

#include <thrust/memory.h>
//#include <thrust/pair.h>
//#include <thrust/random.h>
//#include <thrust/reduce.h>
#include <thrust/sequence.h>
//#include <thrust/sort.h>
#include <thrust/scan.h>
#include <thrust/unique.h>
//#include <thrust/tuple.h>
//#include <thrust/iterator/zip_iterator.h>
//#include <thrust/iterator/constant_iterator.h>
//#include <thrust/iterator/counting_iterator.h>
#include <thrust/system/cuda/execution_policy.h>

#include "hip/hip_vector_types.h"
#include <cstdio>

#include <algorithm>
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <vector>
#include <set>
#include "CImg.h"
using namespace cimg_library;

#include <iostream>
#include <fstream>
#include <string>

#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_vector_types.h>


const int N = 16; 
const int blocksize = 16; 


__global__ 
void hello(char *a, int *b) {

	a[threadIdx.x] += b[threadIdx.x];
}

using namespace std;

int main(){

	string str = " ... ";
	int num = 5;
	str += to_string(num);
	cout<<"str: "<<str<<endl;

	volatile float4 f4;
	volatile int2 i2;
	
	char a[N] = "siii \0\0\0\0\0\0";
	int b[N] = {15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
 
	char *ad;
	int *bd;
	const int csize = N*sizeof(char);
	const int isize = N*sizeof(int);
 
	printf("%s", a);
 
	hipMalloc( (void**)&ad, csize ); 
	hipMalloc( (void**)&bd, isize ); 
	hipMemcpy( ad, a, csize, hipMemcpyHostToDevice ); 
	hipMemcpy( bd, b, isize, hipMemcpyHostToDevice ); 
	
	dim3 dimBlock( blocksize, 1 );
	dim3 dimGrid( 1, 1 );
	hello<<<dimGrid, dimBlock>>>(ad, bd);
	hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost ); 
	hipFree( ad );
	hipFree( bd );
	
	printf("%s\n", a);
	return EXIT_SUCCESS;
}

